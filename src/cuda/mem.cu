#include <tannic/runtime/streams.h>
#include "cuda/exc.cuh"
#include "cuda/mem.cuh" 

namespace cuda::nn { 

// TODO: Add backend api in base library to avoid code repetition here.
  
int getDeviceCount() {
    int count;
    hipError_t err = hipGetDeviceCount(&count); CUDA_CHECK(err);
    return count;
}

void setDevice(int id) {
    CUDA_CHECK(hipSetDevice(id));
}

void* allocate(const device_t* resource, size_t nbytes) {
    setDevice(resource->id); 
    void* ptr = nullptr;
    if (resource->traits & SYNC) { 
        CUDA_CHECK(hipMalloc(&ptr, nbytes));
    } else {
        stream_t stream = pop_stream(resource); 
        hipStream_t cudaStream = reinterpret_cast<hipStream_t>(stream.address);
        CUDA_CHECK(hipMallocAsync(&ptr, nbytes, cudaStream));
        put_stream(resource, stream); 
    }
    return ptr;
} 

void* deallocate(const device_t* resource, void* ptr) {
    setDevice(resource->id);
    if (resource->traits & SYNC) {
        CUDA_CHECK(hipFree(ptr));
    } else {
        stream_t stream = pop_stream(resource); 
        hipStream_t cudaStream = reinterpret_cast<hipStream_t>(stream.address);
        CUDA_CHECK(hipFreeAsync(ptr, cudaStream));
        put_stream(resource, stream); 
    }
    return nullptr;
}

void copyFromHost(const device_t* resource, const void* src , void* dst, size_t nbytes) {
    setDevice(resource->id);
    if (resource->traits & SYNC) {
        hipMemcpy(dst, src, nbytes, hipMemcpyHostToDevice);
    } 
    else {
        stream_t stream = pop_stream(resource); 
        hipStream_t cudaStream = reinterpret_cast<hipStream_t>(stream.address);
        hipMemcpyAsync(dst, src, nbytes, hipMemcpyHostToDevice, cudaStream);
        put_stream(resource, stream); 
    }
} 

bool compareFromHost(const device_t* resource, const void* hst_ptr, const void* dvc_ptr, size_t nbytes) {  
    void* buffer = malloc(nbytes); 
    CUDA_CHECK(hipMemcpy(buffer, dvc_ptr, nbytes, hipMemcpyDeviceToHost));
    bool result = (memcmp(hst_ptr, buffer, nbytes) == 0);
    free(buffer);   
    return result;
}

} // namespace cuda  